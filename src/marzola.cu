#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>
#include <random>
#include <hip/hip_runtime.h>
#include "point_gpu.hpp"
#include "common.hpp"

using namespace std;

__constant__ float d_hyperplanes[DIMENSIONS * N_HYPERPLANES];   // pointer to random hyperplanes stored on device
__constant__ unsigned long long int prime;  // 64-bit prime often used in hashing

/**
 * @brief hahses a signature to an unsigned int representing whether the key is left/right of each hyperplane
 * @param key is the signature
 * @return the hashed signature
 * @note __forceinline__ strongly suggests to inline this function due to its simplicity
 */
__device__ __forceinline__ unsigned int hash_signature(unsigned long long int key) {
    key = (key ^ (key >> 30)) * prime; // Mix upper and lower bits
    key = (key ^ (key >> 27)) * prime; // Further mix bits
    key = key ^ (key >> 31);           // Final mixing
    return static_cast<unsigned int>(key); // Cast to unsigned int (lower 32 bits)
}

/**
 * @brief double the bucket capacity
 * @param bucket is a pointer to the bucket
 * @param current_size is the current size of the bucket
 */
__device__ void resize(int **bucket, unsigned int current_size) {
    // allocate new bucket with twice the size as the old one (requires compute capability of 3.0 or above)
    int *new_bucket = (int*) malloc(current_size * 2 * sizeof(int));

    // copy data from old bucket to new bucket
    for(int i = 0; i < current_size; i++)
        new_bucket[i] = (*bucket)[i];
    
    free(*bucket);   // free old bucket memory
    *bucket = new_bucket; // bucket now points to the new bucket data
}

__device__ inline bool is_power_of_two(unsigned int n) {
    return (n & (n - 1)) == 0;
}

__device__ unsigned long long int signature_gpu(float* points, int point_number) {
    unsigned long long int sig = 0;
    unsigned int exponent = 0;
    float base = 0.0;

    for (int i = 0; i < N_HYPERPLANES; i++) {
        base = 0.0;
        #pragma unroll // when DIMENSIONS IS SMALL
        for (int j = 0; j < DIMENSIONS; j++)
            // base += points[point_number + j] * hyperplanes[DIMENSIONS * i + j];
            base = __fmaf_rn(points[point_number + j], d_hyperplanes[DIMENSIONS * i + j], base);

        if (base >= 0.0) 
            sig += (1ULL << exponent);

        exponent++;
    }   
    return sig;
}

__global__ void add_device(float *points, int **buckets, unsigned long long int *signatures, unsigned int *curr_bucket_used, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_count = gridDim.x * blockDim.x;

    for (int i = tid; i < n; i += thread_count) {
        signatures[i] = signature_gpu(points, i);  // calculate point signature, i.e. whether it is left/right of each hyperplane
        unsigned int bucket_index = (hash_signature(signatures[i]) % (n / thread_count)) + n / thread_count * tid ; // bucket index indicates in which bucket to put this point. Note that we store the index of the point rather than the point itself
        
        if (curr_bucket_used[bucket_index] >= BUCKET_SIZE && is_power_of_two(curr_bucket_used[bucket_index])) { // each time a bucket is full we double its capacity
            resize(&buckets[bucket_index], curr_bucket_used[bucket_index]);
        }

        buckets[bucket_index][curr_bucket_used[bucket_index]] = i; // put the index of the i-th point in the right bucket
        curr_bucket_used[bucket_index]++;
    }
}


/**
 * @brief search n points in the index
 * @param points is the set of points to search
 * @param buckets is the bucket indexes
 * @param result are the resulting indexes
 * @param n the number of points to search
 */
__global__ void search(float *points, int **buckets, unsigned int *bucket_size, unsigned int* result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_count = gridDim.x * blockDim.x;

    for (int i = tid; i < n; i += thread_count) {
        unsigned long long int signature = signature_gpu(points, i);
        result[i] = hash_signature(signature);
    }
}

float *generate_random_hyperplanes(int d, int nbits) {
    float *hyperplanes = new float[d * nbits];

    srand(time(NULL));
    const float lower_bound = -1.0;
    const float upper_bound = 1.0;

    for (int i = 0; i < nbits; i++) {
        for (int j = 0; j < d; j++) {
            hyperplanes[i * d + j] = lower_bound + (upper_bound - lower_bound) * ((float) rand() / RAND_MAX);
        }
    }
    return hyperplanes;
}

float* generate_random_points() {
    float * points = new float[N * DIMENSIONS];
    
    srand(time(NULL));
    const float lower_bound = -1000.0;
    const float upper_bound = 1000.0;
    
    #pragma omp parallel num_threads(4)
    {
        #pragma omp parallel for
        for (int i = 0; i < N * DIMENSIONS; i++) {
            points[i] = lower_bound + (upper_bound - lower_bound) * ((float) rand() / RAND_MAX);
        }
    }
    return points;
}

void allocateMatrixOnDevice(int*** d_matrix, int rows, int cols) {
    int** h_row_ptrs = new int*[rows]; // Host array of pointers

    // Allocate device memory for the array of pointers
    int** d_row_ptrs;
    hipMalloc((void**)&d_row_ptrs, rows * sizeof(int*));

    // Allocate memory for each row on the device
    for (int i = 0; i < rows; ++i) {
        int* d_row;
        hipMalloc((void**)&d_row, cols * sizeof(int));
        h_row_ptrs[i] = d_row; // Store row pointer in host array
    }

    // Copy the row pointers from host to device
    hipMemcpy(d_row_ptrs, h_row_ptrs, rows * sizeof(int*), hipMemcpyHostToDevice);

    // Copy the device row pointers back to the device
    for (int i = 0; i < rows; ++i) {
        hipMemcpy(&(d_row_ptrs[i]), &h_row_ptrs[i], sizeof(int*), hipMemcpyHostToDevice);
    }

    *d_matrix = d_row_ptrs; // Assign the allocated device matrix
    delete[] h_row_ptrs; // Free host row pointers
}

int main() {
    /* POINTERS TO HOST */
    float *h_hyperplanes = generate_random_hyperplanes(DIMENSIONS, N_HYPERPLANES); // host random hyperplanes
    float* h_points = generate_random_points(); // host points
    const unsigned long long int h_prime = 0x9e3779b97f4a7c15ULL;
    /* POINTERS TO HOST */

    /* POINTERS TO DEVICE */
    int **d_buckets;                        // buckets containing the indexes of the points
    unsigned long long int *d_signatures;   // signatures of each point
    unsigned int *d_bucket_size;            // currently used size of each bucket
    float *d_points;                        // points stored on the device
    /* POINTERS TO DEVICE*/

    /* ALLOCATE MEMORY ON DEVICE */
    CHECK_CUDA(hipFree(0)); // Reset the device

    // upload points to device global memory
    CHECK_CUDA(hipMalloc((void**)&d_points, N * DIMENSIONS * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_points, h_points, N * DIMENSIONS * sizeof(float), hipMemcpyHostToDevice));

    // allocate memory for signatures and bucket size on device global memory
    CHECK_CUDA(hipMalloc(&d_signatures, N * sizeof(unsigned long long int)));
    CHECK_CUDA(hipMalloc(&d_bucket_size, N * sizeof(int)));
    CHECK_CUDA(hipMemset(d_bucket_size, 0, N * sizeof(int))); // set all buckets size to 0

    allocateMatrixOnDevice(&d_buckets, N, BUCKET_SIZE);

    // upload hyperplanes to device constant memory
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(d_hyperplanes), h_hyperplanes, sizeof(float) * DIMENSIONS * N_HYPERPLANES));
    // upload prime to constant memory
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(prime), &h_prime, sizeof(unsigned long long int)));

    // increase heap size to allow for dinamic allocation
    size_t heapSize = 1024 * 1024 * 1024; // 1 GB
    CHECK_CUDA(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));

    int number_of_blocks = 32, threads_per_block = 64;
    int n = N;

    add_device<<<number_of_blocks, threads_per_block>>>(d_points, d_buckets, d_signatures, d_bucket_size, n);
    
    CHECK_CUDA(hipDeviceSynchronize());

    // free memory
    CHECK_CUDA(hipFree(d_points));
    CHECK_CUDA(hipFree(d_signatures));
    CHECK_CUDA(hipFree(d_bucket_size));
    CHECK_CUDA(hipFree(d_buckets));

    printf("DONE\n");
    fflush(stdout);
    return 0;   
}