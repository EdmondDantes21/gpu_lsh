#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include "point_gpu.hpp"
#include "common.hpp"

using namespace std;

__constant__ float d_hyperplanes[DIMENSIONS * N_HYPERPLANES];   // pointer to random hyperplanes stored on device
__constant__ unsigned long long int prime;  // 64-bit prime often used in hashing

/**
 * @brief hahses a signature to an unsigned int representing whether the key is left/right of each hyperplane
 * @param key is the signature
 * @return the hashed signature
 * @note __forceinline__ strongly suggests to inline this function due to its simplicity
 */
__device__ __forceinline__ unsigned int hash_signature(unsigned long long int key) {
    key = (key ^ (key >> 30)) * prime; // Mix upper and lower bits
    key = (key ^ (key >> 27)) * prime; // Further mix bits
    key = key ^ (key >> 31);           // Final mixing
    return static_cast<unsigned int>(key); // Cast to unsigned int (lower 32 bits)
}

/**
 * @brief twice the bucket capacity
 * @param bucket is a pointer to the bucket
 * @param current_size is the current size of the bucket
 */
__device__ void resize(int **bucket, int current_size) {
    // allocate new bucket with twice the size as the old one (requires compute capability of 3.0 or above)
    int *new_bucket = (int*) malloc(current_size * 2 * sizeof(int));

    // copy data from old bucket to new bucket
    for(int i = 0; i < current_size; i++)
        new_bucket[i] = *(bucket)[i];
    
    hipFree(*bucket);   // free old bucket memory
    *bucket = new_bucket; // bucket now points to the new bucket data
}

__device__ inline bool is_power_of_two(unsigned int n) {
    return (n & (n - 1)) == 0;
}

__device__ unsigned long long int signature_gpu(float* points, int point_number) {
    unsigned long long int sig = 0;
    unsigned int exponent = 0;
    float base = 0.0;

    for (int i = 0; i < N_HYPERPLANES; i++) {
        base = 0.0;
        #pragma unroll // when DIMENSIONS IS SMALL
        for (int j = 0; j < DIMENSIONS; j++)
            // base += points[point_number + j] * hyperplanes[DIMENSIONS * i + j];
            base = __fmaf_rn(points[point_number + j], d_hyperplanes[DIMENSIONS * i + j], base);

        if (base >= 0.0) 
            sig += (1ULL << exponent);

        exponent++;
    }   
    return sig;
}

__global__ void add_device(float *points, int **buckets, unsigned long long int *signatures, unsigned int *bucket_size, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_count = gridDim.x * blockDim.x;
    
    for (int i = tid; i < n; i += thread_count) {
        signatures[i] = signature_gpu(points, i);  // calculate point signature, i.e. whether it is left/right of each hyperplane
        unsigned int bucket_index = (hash_signature(signatures[i]) & ((n-1) / thread_count)) + n / thread_count * tid ; // bucket index indicates in which bucket to put this point. Note that we store the index of the point rather than the point itself
    
        if (bucket_size[bucket_index] >= BUCKET_SIZE && is_power_of_two(bucket_size[bucket_index])) // each time a bucket is full we double its capacity
            resize(&buckets[bucket_index], bucket_size[bucket_index]);
        buckets[bucket_index][bucket_size[bucket_index]] = i; // put the index of the i-th point in the right bucket
        bucket_size[bucket_index]++;
    }
}

/**
 * @brief search n points in the index
 * @param points is the set of points to search
 * @param buckets is the bucket indexes
 * @param result are the resulting indexes
 * @param n the number of points to search
 */
__global__ void search(float *points, int **buckets, unsigned int *bucket_size, unsigned int* result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_count = gridDim.x * blockDim.x;

    for (int i = tid; i < n; i += thread_count) {
        unsigned long long int signature = signature_gpu(points, i);
        result[i] = hash_signature(signature);
    }
}

float *generate_random_hyperplanes(int d, int nbits) {
    float *hyperplanes = new float[d * nbits];

    srand(time(NULL));
    const float lower_bound = -1.0;
    const float upper_bound = 1.0;

    for (int i = 0; i < nbits; i++) {
        for (int j = 0; j < d; j++) {
            hyperplanes[i * d + j] = lower_bound + (upper_bound - lower_bound) * ((float) rand() / RAND_MAX);
        }
    }
    return hyperplanes;
}

float* generate_random_points() {
    float * points = new float[N * DIMENSIONS];

    srand(time(NULL));
    const float lower_bound = -1000.0;
    const float upper_bound = 1000.0;
    
    for (int i = 0; i < N * DIMENSIONS; i++) {
        points[i] = lower_bound + (upper_bound - lower_bound) * ((float) rand() / RAND_MAX);
    }
    return points;
}

void allocateMatrixOnDevice(int*** d_matrix, int rows, int cols) {
    int** h_row_ptrs = new int*[rows]; // Host array of pointers

    // Allocate device memory for the array of pointers
    int** d_row_ptrs;
    hipMalloc((void**)&d_row_ptrs, rows * sizeof(int*));

    // Allocate memory for each row on the device
    for (int i = 0; i < rows; ++i) {
        int* d_row;
        hipMalloc((void**)&d_row, cols * sizeof(int));
        h_row_ptrs[i] = d_row; // Store row pointer in host array
    }

    // Copy the row pointers from host to device
    hipMemcpy(d_row_ptrs, h_row_ptrs, rows * sizeof(int*), hipMemcpyHostToDevice);

    // Copy the device row pointers back to the device
    for (int i = 0; i < rows; ++i) {
        hipMemcpy(&(d_row_ptrs[i]), &h_row_ptrs[i], sizeof(int*), hipMemcpyHostToDevice);
    }

    *d_matrix = d_row_ptrs; // Assign the allocated device matrix
    delete[] h_row_ptrs; // Free host row pointers
}

int main() {
    /* POINTERS TO HOST */
    float *h_hyperplanes = generate_random_hyperplanes(DIMENSIONS, N_HYPERPLANES); // host random hyperplanes
    float* h_points = generate_random_points(); // host points
    const unsigned long long int h_prime = 0x9e3779b97f4a7c15ULL;
    /* POINTERS TO HOST */

    /* POINTERS TO DEVICE */
    int **d_buckets; // buckets containing the indexes of the points
    unsigned long long int *d_signatures;              // signatures of each point
    unsigned int *d_bucket_size;             // currently used size of each bucket
    //float* d_hyperplanes;           // points to the random hyperplanes on device
    float *d_points;                // points stored on the device
    /* POINTERS TO DEVICE*/

    /* ALLOCATE MEMORY ON DEVICE */
    hipFree(0);

    // upload points to device global memory
    hipMalloc((void**)&d_points, N * DIMENSIONS * sizeof(float));
    hipMemcpy(d_points, h_points, N * DIMENSIONS * sizeof(float), hipMemcpyHostToDevice);

    // allocate memory for signatures and bucket size on device global memory
    hipMalloc(&d_signatures, N * sizeof(unsigned long long int));
    hipMalloc(&d_bucket_size, N * sizeof(int));
    hipMemset(&d_bucket_size, 0, N * sizeof(int)); // set all buckets size to 0

    allocateMatrixOnDevice(&d_buckets, N, BUCKET_SIZE);

    // upload hyperplanes to device constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(d_hyperplanes), h_hyperplanes, sizeof(float) * DIMENSIONS * N_HYPERPLANES);
    // upload prime to constant memory
    hipMemcpyToSymbol(HIP_SYMBOL(prime), &h_prime, sizeof(unsigned long long int));

    int number_of_blocks = 1, threads_per_block = 4;
    int n = 1 << 14;

    add_device<<<number_of_blocks, threads_per_block>>>(d_points, d_buckets, d_signatures, d_bucket_size, n);
    hipDeviceSynchronize();
    printf("done\n");
}