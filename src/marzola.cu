#include "hip/hip_runtime.h"
#include <iostream>
#include <omp.h>
#include <random>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include "point_gpu.hpp"
#include "common.hpp"

using namespace std;

__constant__ float d_hyperplanes[DIMENSIONS * N_HYPERPLANES];   // pointer to random hyperplanes stored on device
__constant__ unsigned long long int prime;  // 64-bit prime often used in hashing

/**
 * @brief hahses a signature to an unsigned int representing whether the key is left/right of each hyperplane
 * @param key is the signature
 * @return the hashed signature
 * @note __forceinline__ strongly suggests to inline this function due to its simplicity
 */
__device__ __forceinline__ unsigned int hash_signature(unsigned long long int key) {
    key = (key ^ (key >> 30)) * prime; // Mix upper and lower bits
    key = (key ^ (key >> 27)) * prime; // Further mix bits
    key = key ^ (key >> 31);           // Final mixing
    return static_cast<unsigned int>(key); // Cast to unsigned int (lower 32 bits)
}

/**
 * @brief double the bucket capacity
 * @param bucket is a pointer to the bucket
 * @param current_size is the current size of the bucket
 */
__device__ void resize(int **bucket, unsigned int current_size) {
    // allocate new bucket with twice the size as the old one (requires compute capability of 3.0 or above)
    int *new_bucket = (int*) malloc(current_size * 2 * sizeof(int));

    // if (new_bucket == NULL) {
    //     printf("Failed to allocate memory on the device\n");
    //     return;
    // }

    // copy data from old bucket to new bucket
    for(int i = 0; i < current_size; i++)
        new_bucket[i] = (*bucket)[i];
    
    free(*bucket);   // free old bucket memory
    *bucket = new_bucket; // bucket now points to the new bucket data
}

__device__ inline bool is_power_of_two(unsigned int n) {
    return (n & (n - 1)) == 0;
}

__device__ unsigned long long int signature_gpu(float* points, int point_number) {
    unsigned long long int sig = 0;
    unsigned int exponent = 0;
    float base = 0.0;

    for (int i = 0; i < N_HYPERPLANES; i++) {
        base = 0.0;
        #pragma unroll // when DIMENSIONS IS SMALL
        for (int j = 0; j < DIMENSIONS; j++)
            base = __fmaf_rn(points[point_number + j], d_hyperplanes[DIMENSIONS * i + j], base);

        if (base >= 0.0) 
            sig += (1ULL << exponent);

        exponent++;
    }   
    return sig;
}

__global__ void add_device(float *points, int **buckets, unsigned long long int *signatures, unsigned int *curr_bucket_used, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_count = gridDim.x * blockDim.x;

    for (int i = tid; i < n; i += thread_count) {
        signatures[i] = signature_gpu(points, i);  // calculate point signature, i.e. whether it is left/right of each hyperplane
        unsigned int bucket_index = (hash_signature(signatures[i]) % (n / thread_count)) + n / thread_count * tid ; // bucket index indicates in which bucket to put this point. Note that we store the index of the point rather than the point itself
        
        if (curr_bucket_used[bucket_index] >= BUCKET_SIZE && is_power_of_two(curr_bucket_used[bucket_index])) { // each time a bucket is full we double its capacity
            resize(&buckets[bucket_index], curr_bucket_used[bucket_index]);
        }

        buckets[bucket_index][curr_bucket_used[bucket_index]] = i; // put the index of the i-th point in the right bucket
        curr_bucket_used[bucket_index]++;
    }
}


/**
 * @brief search n points in the index
 * @param points is the set of points to search
 * @param buckets is the bucket indexes
 * @param result are the resulting indexes
 * @param n the number of points to search
 */
__global__ void search(float *points, int **buckets, unsigned int *bucket_size, unsigned int* result, int n) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int thread_count = gridDim.x * blockDim.x;

    for (int i = tid; i < n; i += thread_count) {
        unsigned long long int signature = signature_gpu(points, i);
        result[i] = hash_signature(signature);
    }
}

float *generate_random_hyperplanes(int d, int nbits) {
    float *hyperplanes = new float[d * nbits];

    srand(time(NULL));
    const float lower_bound = -1.0;
    const float upper_bound = 1.0;

    for (int i = 0; i < nbits; i++) {
        for (int j = 0; j < d; j++) {
            hyperplanes[i * d + j] = lower_bound + (upper_bound - lower_bound) * ((float) rand() / RAND_MAX);
        }
    }
    return hyperplanes;
}

float* generate_random_points() {
    float * points = new float[N * DIMENSIONS];
    
    srand(time(NULL));
    const float lower_bound = -1000.0;
    const float upper_bound = 1000.0;
    
    #pragma omp parallel num_threads(4)
    {
        #pragma omp parallel for
        for (int i = 0; i < N * DIMENSIONS; i++) {
            points[i] = lower_bound + (upper_bound - lower_bound) * ((float) rand() / RAND_MAX);
        }
    }
    return points;
}

void allocateMatrixOnDevice(int*** d_matrix, int rows, int cols, hipStream_t stream) {
    int** h_row_ptrs = new int*[rows]; // Host array of pointers

    // Allocate device memory for the array of pointers
    int** d_row_ptrs;
    CHECK_CUDA(hipMallocAsync((void**)&d_row_ptrs, rows * sizeof(int*), stream));

    // Allocate memory for each row on the device
    for (int i = 0; i < rows; ++i) {
        int* d_row;
        CHECK_CUDA(hipMallocAsync((void**)&d_row, cols * sizeof(int), stream));
        h_row_ptrs[i] = d_row; // Store row pointer in host array
    }

    // Copy the row pointers from host to device
    CHECK_CUDA(hipMemcpyAsync(d_row_ptrs, h_row_ptrs, rows * sizeof(int*), hipMemcpyHostToDevice, stream));

    // Copy the device row pointers back to the device
    for (int i = 0; i < rows; ++i) {
        CHECK_CUDA(hipMemcpyAsync(&(d_row_ptrs[i]), &h_row_ptrs[i], sizeof(int*), hipMemcpyHostToDevice, stream));
    }

    *d_matrix = d_row_ptrs; // Assign the allocated device matrix
    delete[] h_row_ptrs; // Free host row pointers
}

int main() {
    /* PARAMETERS */
    int number_of_blocks = 96, threads_per_block = 96;
    int n = N;

    /* POINTERS TO HOST */
    float *h_hyperplanes = generate_random_hyperplanes(DIMENSIONS, N_HYPERPLANES); // host random hyperplanes
    float* h_points = generate_random_points(); // host points
    const unsigned long long int h_prime = 0x9e3779b97f4a7c15ULL;
    /* POINTERS TO HOST */

    /* POINTERS TO DEVICE */
    hipStream_t stream1, stream2, stream3; // CUDA streams for asynchronous operations
    int **d_buckets;                        // buckets containing the indexes of the points
    unsigned long long int *d_signatures;   // signatures of each point
    unsigned int *d_bucket_size;            // currently used size of each bucket
    float *d_points;                        // points stored on the device
    /* POINTERS TO DEVICE*/

    struct timeval start, end;
    gettimeofday(&start, NULL);

    /* ALLOCATE MEMORY ON DEVICE */
    CHECK_CUDA(hipStreamCreate(&stream1));
    CHECK_CUDA(hipStreamCreate(&stream2));
    CHECK_CUDA(hipStreamCreate(&stream3));

    // Upload points to device global memory (asynchronously)
    CHECK_CUDA(hipMallocAsync((void**)&d_points, N * DIMENSIONS * sizeof(float), stream1));
    CHECK_CUDA(hipMemcpyAsync(d_points, h_points, N * DIMENSIONS * sizeof(float), hipMemcpyHostToDevice, stream1));

    // Allocate memory for signatures and bucket size on device global memory (asynchronously)
    CHECK_CUDA(hipMallocAsync(&d_signatures, N * sizeof(unsigned long long int), stream2));
    CHECK_CUDA(hipMallocAsync(&d_bucket_size, N * sizeof(int), stream2));
    CHECK_CUDA(hipMemsetAsync(d_bucket_size, 0, N * sizeof(int), stream2));  // Set all bucket sizes to 0 asynchronously

    // Upload hyperplanes and prime constant to device constant memory asynchronously
    CHECK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_hyperplanes), h_hyperplanes, sizeof(float) * DIMENSIONS * N_HYPERPLANES, 0, hipMemcpyHostToDevice, stream1));
    CHECK_CUDA(hipMemcpyToSymbolAsync(HIP_SYMBOL(prime), &h_prime, sizeof(unsigned long long int), 0, hipMemcpyHostToDevice, stream1));

    allocateMatrixOnDevice(&d_buckets, N, BUCKET_SIZE, stream3);

    // increase heap size to allow for dinamic allocation
    size_t heapSize = 1024 * 1024 * 1024; // 1 GB
    CHECK_CUDA(hipDeviceSetLimit(hipLimitMallocHeapSize, heapSize));

    // Ensure all memory transfers and allocations complete before launching kernel
    CHECK_CUDA(hipStreamSynchronize(stream1));
    CHECK_CUDA(hipStreamSynchronize(stream2));
    CHECK_CUDA(hipStreamSynchronize(stream3));

    add_device<<<number_of_blocks, threads_per_block, 0, stream1>>>(d_points, d_buckets, d_signatures, d_bucket_size, n);

    hipDeviceSynchronize();
    
    gettimeofday(&end, NULL);
    long long int time_usec = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec));
    cout << "execution time = " << time_usec / 1000000.0 << endl;

    // Reset device
    CHECK_CUDA(hipDeviceReset());

    return 0;
}